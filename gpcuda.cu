#include "hip/hip_runtime.h"
#include "cuda-common.hxx"

typedef double real_t;

__global__ void cos_kernel(const real_t * in, real_t * out,
                           size_t size) {
    unsigned ii = blockIdx.x * blockDim.x + threadIdx.x;
    if(ii < size) {
        out[ii] = cos(in[ii]);
    }
}

extern "C" {
    void cos_doubles(double *in_array, double *out_array, int size) {
        
        real_t *d_in, *d_out;
        
        CUDA_CALL(hipMalloc((void **) &d_in, sizeof(real_t)*size));
        CUDA_CALL(hipMalloc((void **) &d_out, sizeof(real_t)*size));        

        CUDA_CALL(hipMemcpy(d_in, in_array, sizeof(real_t)*size,
                             hipMemcpyHostToDevice));

        cos_kernel<<<ceil(size/1024.0), 1024>>>(d_in, d_out, size);
        CUDA_CALL(hipPeekAtLastError());
        CUDA_CALL(hipDeviceSynchronize());

        CUDA_CALL(hipMemcpy(out_array, d_out, sizeof(real_t)*size,
                             hipMemcpyDeviceToHost));

        CUDA_CALL(hipFree(d_in));
        CUDA_CALL(hipFree(d_out));
                  
    }
}
